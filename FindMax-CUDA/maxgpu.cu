#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

__global__ void findMaxKernel(unsigned int *array, unsigned int *max, int *mutex, unsigned int n)
{
	unsigned int index = threadIdx.x + blockIdx.x*blockDim.x;
	unsigned int stride = gridDim.x*blockDim.x;
	unsigned int offset = 0;

	__shared__ unsigned int cache[1024];

	unsigned int temp = 0;
	while(index + offset < n){
		temp = (temp > array[index + offset]) ? temp : array[index+offset];
		offset += stride;
	}

	cache[threadIdx.x] = temp;

	__syncthreads();

	// reduction
	unsigned int i = blockDim.x/2;
	while(i != 0){
		if(threadIdx.x < i){
			cache[threadIdx.x] = (cache[threadIdx.x] > cache[threadIdx.x + i]) ? cache[threadIdx.x] : cache[threadIdx.x + i];
		}

		__syncthreads();
		i /= 2;
	}

	if(threadIdx.x == 0){		
		atomicMax(max, cache[0]);
	}
}

unsigned int getmaxcu(unsigned int num[], unsigned int size)
{
	unsigned int *d_num;
	unsigned int *h_max;
	unsigned int *d_max;
	int *d_mutex;

	//Allocate memory
	h_max = (unsigned int*)malloc(sizeof(unsigned int));
	hipMalloc((void**)&d_num, size*sizeof(unsigned int));
	hipMalloc((void**)&d_max, sizeof(unsigned int));
	hipMalloc((void**)&d_mutex, sizeof(int));
	hipMemset(d_max, 0, sizeof(unsigned int));
	hipMemset(d_mutex, 0, sizeof(unsigned int));

	//Copy from host to device
	hipMemcpy(d_num, num, size*sizeof(unsigned int), hipMemcpyHostToDevice);

	// call kernel
	dim3 gridSize = 256;
	dim3 blockSize = 1024;
	findMaxKernel<<< gridSize, blockSize >>>(d_num, d_max, d_mutex, size);

	//Copy from device to host
	hipMemcpy(h_max, d_max, sizeof(unsigned int), hipMemcpyDeviceToHost);

	// free memory	
	hipFree(d_num);
	hipFree(d_max);
	hipFree(d_mutex);

	return h_max[0];
}

int main(int argc, char *argv[])
{
    unsigned int size = 0;  // The size of the array
    unsigned int i;  // loop index
    unsigned int * numbers; //pointer to the array
    
    if(argc !=2)
    {
		printf("usage: maxseq num\n");
		printf("num = size of the array\n");
		exit(1);
    }
   
    size = atol(argv[1]);

    numbers = (unsigned int *)malloc(size * sizeof(unsigned int));
    if( !numbers )
    {
		printf("Unable to allocate mem for an array of size %u\n", size);
		exit(1);
    }    

	srand(time(NULL)); // setting a seed for the random number generator
    // Fill-up the array with random numbers from 0 to size-1 
    for( i = 0; i < size; i++)
		numbers[i] = rand()  % size;    
   
    printf(" The maximum number in the array is: %u\n", getmaxcu(numbers, size));

	free(numbers);
	exit(0);
}
